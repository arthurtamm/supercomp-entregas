#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>   // Inclui a biblioteca Thrust para vetores na GPU
#include <thrust/host_vector.h>     // Inclui a biblioteca Thrust para vetores na CPU
#include <iostream>                 // Biblioteca padrão para entrada e saída de dados
#include <chrono>                   // Biblioteca para medir o tempo de execução do código
#include <thrust/extrema.h> 
using namespace std;

int main() {
    int n = 3112;                   // Define o tamanho do vetor (2518 elementos)
    double value = 0.0;             // Variável auxiliar para armazenar o valor lido da entrada

    // Cria um vetor na CPU (host_vector) com n elementos, todos inicializados com 0
    thrust::host_vector<double> host(n, 0);

    // Loop para ler n valores da entrada padrão e armazená-los no vetor host
    for (int i = 0; i < n; i++) {
        cin >> value;              // Lê um valor da entrada padrão
        host[i] = value;           // Armazena o valor lido no vetor host
    }

    // Cria um vetor na GPU (device_vector) copiando os dados do vetor host da CPU
    thrust::device_vector<double> dev(host);

    double sum10 = thrust::reduce(dev.begin(), dev.end(), 0, thrust::plus<double>());
    double average10 = sum10 / n;
    std::cout << "Average last 10 years: " << average10 << std::endl;

    double sum1 = thrust::reduce(dev.end()-365, dev.end(), 0, thrust::plus<double>());
    double average1 = sum1 / 365;
    std::cout << "Average last 1 year: " << average1 << std::endl;

    double min = *thrust::min_element(dev.begin(), dev.end());
    std::cout << "Minimum value 10 years: " << min << std::endl;

    double max = *thrust::max_element(dev.begin(), dev.end());
    std::cout << "Maximum value 10 years: " << max << std::endl;

    double min1 = *thrust::min_element(dev.end()-365, dev.end());
    std::cout << "Minimum value last year: " << min1 << std::endl;

    double max1 = *thrust::max_element(dev.end()-365, dev.end());
    std::cout << "Maximum value last year: " << max1 << std::endl;


    return 0;
}