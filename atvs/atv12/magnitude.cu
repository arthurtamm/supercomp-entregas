#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>   // Inclui a biblioteca Thrust para vetores na GPU
#include <thrust/host_vector.h>     // Inclui a biblioteca Thrust para vetores na CPU
#include <iostream>                 // Biblioteca padrão para entrada e saída de dados
#include <chrono>  
#include <thrust/sequence.h>
#include <thrust/transform_reduce.h>
using namespace std;
using namespace std::chrono;

struct square
{
    __host__ __device__
    float operator()(const float& x) const {
        return x * x;  // Função de elevação ao quadrado
    }
};

float magnitude(thrust::device_vector<float>& v) {
    float sum_of_squares = thrust::transform_reduce(v.begin(), v.end(), square(), 0.0f, thrust::plus<float>());
    return std::sqrt(sum_of_squares);
}

float magnitude_host(thrust::host_vector<float>& v) {
    float sum_of_squares = thrust::transform_reduce(v.begin(), v.end(), square(), 0.0f, thrust::plus<float>());
    return std::sqrt(sum_of_squares);
}

int main() {

    thrust::host_vector<float> v(20000);

    thrust::fill(v.begin(), v.end(), 2);

    auto start_cpu = high_resolution_clock::now();
    float mag_v = magnitude_host(v);
    auto end_cpu = high_resolution_clock::now();
    duration<double> duration_cpu = end_cpu - start_cpu;

    cout << "Thrust CPU" << endl;
    cout << "Magnitude: " << mag_v << endl;
    cout << "Execution time: " << duration_cpu.count() << "seconds" << endl;

    cout << "---------------------------------" << endl;
    
    thrust::device_vector<float> d_v(v);
    auto start_gpu = high_resolution_clock::now();
    float mag_d_v = magnitude(d_v);
    auto end_gpu = high_resolution_clock::now();
    duration<double> duration_gpu = end_gpu - start_gpu;

    cout << "Thrust GPU" << endl;
    cout << "Magnitude: " << mag_d_v << endl;
    cout << "Execution time: " << duration_gpu.count() << "seconds" << endl;
    


    return 0;
}